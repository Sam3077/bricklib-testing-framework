#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

// $START naive
__global__ void poisson_naive(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    unsigned i = OFF0 + (blockIdx_x) * TILE0 + threadIdx_x;
    unsigned j = OFF1 + (blockIdx_y) * TILE1 + threadIdx_y;
    unsigned k = OFF2 + (blockIdx_z) * TILE2 + threadIdx_z;

    out[k][j][i] = 2.666 * in[k][j][i] - 
        (0.166 * (in[k - 1][j][i] + in[k + 1][j][i] +
                in[k][j - 1][i] + in[k][j + 1][i] +
                in[k][j][i - 1] + in[k][j][i - 1])) -
        (0.0833 * (in[k - 1][j - 1][i] + in[k + 1][j - 1][i] +
                in[k - 1][j + 1][i] + in[k + 1][j + 1][i] +
                in[k - 1][j][i - 1] + in[k + 1][j][i - 1] +
                in[k][j - 1][i - 1] + in[k][j + 1][i - 1] +
                in[k - 1][j][i + 1] + in[k + 1][j][i + 1] +
                in[k][j - 1][i + 1] + in[k][j + 1][i + 1]));
}
// $END naive

// $START codegen
#define in(a, b, c) in_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]
__global__ void poisson_codegen(bElem (*in_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx_z * TILE2);
    long j = OFF1 + (blockIdx_y * TILE1);
    long i = OFF0 + (blockIdx_x * TILE0);
    tile("$PYTHON", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}
#undef in
#undef out
// $END codegen

// $START naive-bricks
__global__ void poisson_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
    unsigned b = grid[blockIdx_z + GB2][blockIdx_y + GB1][blockIdx_x + GB0];
    unsigned i = threadIdx_x;
    unsigned j = threadIdx_y;
    unsigned k = threadIdx_z;

    out[b][k][j][i] = 2.666 * in[b][k][j][i] - 
        (0.166 * (in[b][k - 1][j][i] + in[b][k + 1][j][i] +
                in[b][k][j - 1][i] + in[b][k][j + 1][i] +
                in[b][k][j][i - 1] + in[b][k][j][i - 1])) -
        (0.0833 * (in[b][k - 1][j - 1][i] + in[b][k + 1][j - 1][i] +
                in[b][k - 1][j + 1][i] + in[b][k + 1][j + 1][i] +
                in[b][k - 1][j][i - 1] + in[b][k + 1][j][i - 1] +
                in[b][k][j - 1][i - 1] + in[b][k][j + 1][i - 1] +
                in[b][k - 1][j][i + 1] + in[b][k + 1][j][i + 1] +
                in[b][k][j - 1][i + 1] + in[b][k][j + 1][i + 1]));
}
// $END naive-bricks

// $START codegen-bricks
__global__ void poisson_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType in, BType out) {
  unsigned b = grid[blockIdx_z + GB2][blockIdx_y + GB1][blockIdx_x + GB0];
  brick("$PYTHON", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks