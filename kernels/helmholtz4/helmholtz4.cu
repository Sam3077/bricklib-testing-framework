#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

// $START naive
__global__ void helmholtz4_naive(bElem (*x)[STRIDE1][STRIDE0], bElem (*alpha)[STRIDE1][STRIDE0], 
  bElem (*beta_i)[STRIDE1][STRIDE0], bElem (*beta_j)[STRIDE1][STRIDE0], bElem (*beta_k)[STRIDE1][STRIDE0], 
  bElem (*out)[STRIDE1][STRIDE0], bElem *c) {
    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];

    out[k][j][i] = c1 * alpha[k][j][i] * x[k][j][i]] -
        c2 * h2inv *
        (0.0833 * 
            (beta_i[k][j][i] * (15.0 * (x[k][j][i - 1] - x[k][j][i]) - 
                (x[k][j][i - 1] - x[k][j][i + 1])) + 
            beta_i[k][j][i + 1] * (15.0 * (x[k][j][i + 1] - x[k][j][i]) - 
                (x[k][j][i + 2] - x[k][j][i - 1])) +
            beta_j[k][j][i] * (15.0 * (x[k][j - 1][i] - x[k][j][i]) - 
                (x[k][j - 1][i] - x[k][j + 1][i])) +
            beta_j[k][j + 1][i] * (15.0 * (x[k][j + 1][i] - x[k][j][i]) -
                (x[k][j + 2][i] - x[k][j - 1][i])) +
            beta_k[k][j][i] * (15.0 * (x[k - 1][j][i] - x[k][j][i]) -
                (x[k - 2][j][i] - x[k + 1][j][i])) +
            beta_k[k + 1][j][i] * (15.0 * (x[k + 1][j][i] - x[k][j][i]) -
                (x[k + 2][j][i] - x[k - 1][j][i]))) +

        0.25 * 0.0833 * 
            ((beta_i[k][j + 1][i] - beta_i[k][j - 1][i]) *
                (x[k][j + 1][i - 1] - x[k][j + 1][i] -
                 x[k][j - 1][i - 1] + x[k][j - 1][i]) +
            (beta_i[k + 1][j][i] - beta_i[k - 1][j][i]) * 
                (x[k + 1][j][i - 1] - x[k + 1][j][i] -
                 x[k - 1][j][i - 1] + x[k - 1][j][i]) +
            (beta_j[k][j][i + 1] - beta_j[k][j][i - 1]) *
                (x[k][j - 1][i + 1] - x[k][j][i + 1] -
                 x[k][j - 1][i - 1] + x[k][j][i - 1]) +
            (beta_j[k + 1][j][i] - beta_j[k - 1][j][i]) *
                (x[k + 1][j - 1][i] - x[k + 1][j][i] -
                 x[k - 1][j - 1][i] + x[k - 1][j][i]) +
            (beta_k[k][j][i + 1] - beta_k[k][j][i - 1]) *
                (x[k - 1][j][i + 1] - x[k][j][i + 1] -
                 x[k - 1][j][i - 1] + x[k][j][i - 1]) +
            (beta_k[k][j + 1][i] - beta_k[k][j - 1][i]) *
                (x[k - 1][j + 1][i] - x[k][j + 1][i] -
                 x[k - 1][j - 1][i] + x[k][j - 1][i]) +

            (beta_i[k][j + 1][i + 1] - beta_i[k][j - 1][i + 1]) *
                (x[k][j + 1][i + 1] - x[k][j + 1][i] -
                 x[k][j - 1][i + 1] + x[k][j - 1][i]) + 
            (beta_i[k + 1][j][i + 1] - beta_i[k - 1][j][i + 1]) *
                (x[k + 1][j][i + 1] - x[k + 1][j][i] - 
                 x[k - 1][j][i + 1] + x[k - 1][j][i]) +
            (beta_j[k][j + 1][i + 1] - beta_j[k][j + 1][i - 1]) *
                (x[k][j + 1][i + 1] - x[k][j][i + 1] -
                 x[k][j + 1][i - 1] + x[k][j][i - 1]) +
            (beta_j[k + 1][j + 1][i] - beta_j[k - 1][j + 1][i]) *
                (x[k + 1][j + 1][i] - x[k + 1][j][i] -
                 x[k - 1][j + 1][i] + x[k - 1][j][i]) +
            (beta_k[k + 1][j][i + 1] - beta_k[k + 1][j][i - 1]) *
                (x[k + 1][j][i + 1] - x[k][j][i + 1] -
                 x[k + 1][j][i - 1] + x[k][j][i - 1]) +
            (beta_k[k + 1][j + 1][i] - beta_k[k + 1][j - 1][i]) *
                (x[k + 1][j + 1][i] - x[k][j + 1][i] -
                 x[k + 1][j - 1][i] + x[k][j - 1][i])
        ));
}
// $END naive

// $START codegen
#define x(a, b, c) x_arr[c][b][a]
#define alpha(a, b, c) alpha_arr[c][b][a]
#define beta_i(a, b, c) beta_i_arr[c][b][a]
#define beta_j(a, b, c) beta_j_arr[c][b][a]
#define beta_k(a, b, c) beta_k_arr[c][b][a] 
#define out(a, b, c) out_arr[c][b][a]

__global__ void helmholtz4_codegen(bElem (*x_arr)[STRIDE1][STRIDE0], bElem (*alpha_arr)[STRIDE1][STRIDE0], 
  bElem (*beta_i_arr)[STRIDE1][STRIDE0], bElem (*beta_j_arr)[STRIDE1][STRIDE0], bElem (*beta_k_arr)[STRIDE1][STRIDE0], 
  bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c) {
    unsigned i = OFF0 + (blockIdx_x) * TILE0 + threadIdx_x;
    unsigned j = OFF1 + (blockIdx_y) * TILE1 + threadIdx_y;
    unsigned k = OFF2 + (blockIdx_z) * TILE2 + threadIdx_z;

    tile("$PYTHON", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

#undef x
#undef alpha
#undef beta_i
#undef beta_j
#undef beta_k
#undef out
// $END codegen

// $START naive-bricks
__global__ void helmholtz4_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha,
  BType beta_i, BType beta_j, BType beta_k,
  BType out, bElem *c) {
    unsigned b = grid[blockIdx_z + GB2][blockIdx_y + GB1][blockIdx_x + GB0];
    unsigned i = threadIdx_x;
    unsigned j = threadIdx_y;
    unsigned k = threadIdx_z;

    bElem c1 = c[0];
    bElem c2 = c[1];
    bElem h2inv = c[2];

    out[b][k][j][i] = c1 * alpha[b][k][j][i] * x[b][k][j][i]] -
        c2 * h2inv *
        (0.0833 * 
            (beta_i[b][k][j][i] * (15.0 * (x[b][k][j][i - 1] - x[b][k][j][i]) - 
                (x[b][k][j][i - 1] - x[b][k][j][i + 1])) + 
            beta_i[b][k][j][i + 1] * (15.0 * (x[b][k][j][i + 1] - x[b][k][j][i]) - 
                (x[b][k][j][i + 2] - x[b][k][j][i - 1])) +
            beta_j[b][k][j][i] * (15.0 * (x[b][k][j - 1][i] - x[b][k][j][i]) - 
                (x[b][k][j - 1][i] - x[b][k][j + 1][i])) +
            beta_j[b][k][j + 1][i] * (15.0 * (x[b][k][j + 1][i] - x[b][k][j][i]) -
                (x[b][k][j + 2][i] - x[b][k][j - 1][i])) +
            beta_k[b][k][j][i] * (15.0 * (x[b][k - 1][j][i] - x[b][k][j][i]) -
                (x[b][k - 2][j][i] - x[b][k + 1][j][i])) +
            beta_k[b][k + 1][j][i] * (15.0 * (x[b][k + 1][j][i] - x[b][k][j][i]) -
                (x[b][k + 2][j][i] - x[b][k - 1][j][i]))) +

        0.25 * 0.0833 * 
            ((beta_i[b][k][j + 1][i] - beta_i[b][k][j - 1][i]) *
                (x[b][k][j + 1][i - 1] - x[b][k][j + 1][i] -
                 x[b][k][j - 1][i - 1] + x[b][k][j - 1][i]) +
            (beta_i[b][k + 1][j][i] - beta_i[b][k - 1][j][i]) * 
                (x[b][k + 1][j][i - 1] - x[b][k + 1][j][i] -
                 x[b][k - 1][j][i - 1] + x[b][k - 1][j][i]) +
            (beta_j[b][k][j][i + 1] - beta_j[b][k][j][i - 1]) *
                (x[b][k][j - 1][i + 1] - x[b][k][j][i + 1] -
                 x[b][k][j - 1][i - 1] + x[b][k][j][i - 1]) +
            (beta_j[b][k + 1][j][i] - beta_j[b][k - 1][j][i]) *
                (x[b][k + 1][j - 1][i] - x[b][k + 1][j][i] -
                 x[b][k - 1][j - 1][i] + x[b][k - 1][j][i]) +
            (beta_k[b][k][j][i + 1] - beta_k[b][k][j][i - 1]) *
                (x[b][k - 1][j][i + 1] - x[b][k][j][i + 1] -
                 x[b][k - 1][j][i - 1] + x[b][k][j][i - 1]) +
            (beta_k[b][k][j + 1][i] - beta_k[b][k][j - 1][i]) *
                (x[b][k - 1][j + 1][i] - x[b][k][j + 1][i] -
                 x[b][k - 1][j - 1][i] + x[b][k][j - 1][i]) +

            (beta_i[b][k][j + 1][i + 1] - beta_i[b][k][j - 1][i + 1]) *
                (x[b][k][j + 1][i + 1] - x[b][k][j + 1][i] -
                 x[b][k][j - 1][i + 1] + x[b][k][j - 1][i]) + 
            (beta_i[b][k + 1][j][i + 1] - beta_i[b][k - 1][j][i + 1]) *
                (x[b][k + 1][j][i + 1] - x[b][k + 1][j][i] - 
                 x[b][k - 1][j][i + 1] + x[b][k - 1][j][i]) +
            (beta_j[b][k][j + 1][i + 1] - beta_j[b][k][j + 1][i - 1]) *
                (x[b][k][j + 1][i + 1] - x[b][k][j][i + 1] -
                 x[b][k][j + 1][i - 1] + x[b][k][j][i - 1]) +
            (beta_j[b][k + 1][j + 1][i] - beta_j[b][k - 1][j + 1][i]) *
                (x[b][k + 1][j + 1][i] - x[b][k + 1][j][i] -
                 x[b][k - 1][j + 1][i] + x[b][k - 1][j][i]) +
            (beta_k[b][k + 1][j][i + 1] - beta_k[b][k + 1][j][i - 1]) *
                (x[b][k + 1][j][i + 1] - x[b][k][j][i + 1] -
                 x[b][k + 1][j][i - 1] + x[b][k][j][i - 1]) +
            (beta_k[b][k + 1][j + 1][i] - beta_k[b][k + 1][j - 1][i]) *
                (x[b][k + 1][j + 1][i] - x[b][k][j + 1][i] -
                 x[b][k + 1][j - 1][i] + x[b][k][j - 1][i])
        ));
} 
// $END naive-bricks

// $START codegen-bricks
__global__ void helmholtz4_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType x, BType alpha,
  BType beta_i, BType beta_j, BType beta_k,
  BType out, bElem *c) {
    unsigned b = grid[blockIdx_z + GB2][blockIdx_y + GB1][blockIdx_x + GB0];
    brick("$PYTHON", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks
