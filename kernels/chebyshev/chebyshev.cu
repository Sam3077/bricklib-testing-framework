#include "hip/hip_runtime.h"
#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

// $START naive
__global__ void chebyshev_naive(bElem (*Ac)[STRIDE1][STRIDE0], bElem (*Ap)[STRIDE1][STRIDE0], bElem (*Dinv)[STRIDE1][STRIDE0], bElem (*RHS)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], bElem *c, bElem (*coeff)[4][4]) {
  const size_t radius = $SIZE;
  unsigned i = OFF0 + (blockIdx_x) * TILE0 + threadIdx_x;
  unsigned j = OFF1 + (blockIdx_y) * TILE1 + threadIdx_y;
  unsigned k = OFF2 + (blockIdx_z) * TILE2 + threadIdx_z;

  bElem base = 0;
  #pragma unroll
  for (int i_diff = -radius; i_diff <= radius; i_diff++) {
    #pragma unroll
    for (int j_diff = -radius; j_diff <= radius; j_diff++) {
      #pragma unroll
      for (int k_diff = -radius; k_diff <= radius; k_diff++) {
	base += coeff[i_diff + radius][j_diff + radius][k_diff + radius] * (Ac[i + i_diff][j + j_diff][k + k_diff]);
      }
    }
  }
  out[i][j][k] = Ac[i][j][k] + c[0] * (Ac[i][j][k] + Ap[i][j][k]) + c[1] * Dinv[i][j][k] *
    (RHS[i][j][k] + (Ac[i][j][k] + c[2] * base));
}
// $END naive

// $START naive-bricks
__global__ void chebyshev_naive_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType Ac, BType Ap, BType Dinv, BType RHS, BType out, bElem *c, bElem (*coeff)[4][4]) {
  const size_t radius = $SIZE;
  unsigned b = grid[blockIdx_z + GB2][blockIdx_y + GB1][blockIdx_x + GB0];
  unsigned i = threadIdx_x;
  unsigned j = threadIdx_y;
  unsigned k = threadIdx_z;

  bElem base = 0;
  #pragma unroll
  for (int i_diff = -radius; i_diff <= radius; i_diff++) {
    #pragma unroll
    for (int j_diff = -radius; j_diff <= radius; j_diff++) {
      #pragma unroll
      for (int k_diff = -radius; k_diff <= radius; k_diff++) {
	base += coeff[i_diff + radius][j_diff + radius][k_diff + radius] * (Ac[b][i + i_diff][j + j_diff][k + k_diff]);
      }
    }
  }
  out[b][i][j][k] = Ac[b][i][j][k] + c[0] * (Ac[b][i][j][k] + Ap[b][i][j][k]) + c[1] * Dinv[b][i][j][k] * (RHS[b][i][j][k] + (Ac[b][i][j][k] + c[2] * base));
}
// $END naive-bricks

// $START codegen-bricks
__global__ void chebyshev_codegen_bricks(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType Ac, BType Ap, BType Dinv, BType RHS, BType out, bElem *c, bElem (*coeff)[4][4]) {
  unsigned b = grid[blockIdx_z + GB2][blockIdx_y + GB1][blockIdx_x + GB0];
  brick("$PYTHON", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}
// $END codegen-bricks

// $START codegen
#define Ac(a, b, c) Ac_arr[c][b][a]
#define Ap(a, b, c) Ap_arr[c][b][a]
#define Dinv(a, b, c) Dinv_arr[c][b][a]
#define RHS(a, b, c) RHS_arr[c][b][a]
#define out(a, b, c) out_arr[c][b][a]

__global__ void chebyshev_codegen(bElem (*Ac_arr)[STRIDE1][STRIDE0], bElem (*Ap_arr)[STRIDE1][STRIDE0], bElem (*Dinv_arr)[STRIDE1][STRIDE0], bElem (*RHS_arr)[STRIDE1][STRIDE0], bElem (*out_arr)[STRIDE1][STRIDE0], bElem *c, bElem (*coeff)[4][4]) {
  long k = OFF2 + (blockIdx_z * TILE2);
  long j = OFF1 + (blockIdx_y * TILE1);
  long i = OFF0 + (blockIdx_x * TILE0);
  tile("$PYTHON", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

#undef Ac
#undef Ap
#undef Dinv
#undef RHS
#undef out
// $END codegen

