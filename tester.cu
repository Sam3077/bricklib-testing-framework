#include "hip/hip_runtime.h"
#include <iostream>

#include "bricksetup.h"
#include "multiarray.h"
#include "brickcompare.h"
#include <omp.h>
#include <cmath>
#include <cassert>
#include <string.h>

#include "brick.h"

#include "./gen/script_vars.h"
#include "./gen/consts.h"
#include "./gen/incls.h"

#define gpuSynchronizeAssert() assert(gpuDeviceSynchronize() == gpuSuccess)

const char *get_atype_kernel_name(AFunc ptr) {
    for (int i = 0; i < afunc_count + bfunc_count; i++) {
        if (ptr == func_name_lut[i].a) {
            return func_name_lut[i].funcName;
        }
    }
    return "unknown";
}
const char *get_btype_kernel_name(BFunc ptr) {
    for (int i = 0; i < afunc_count + bfunc_count; i++) {
        if (ptr == func_name_lut[i].b) {
            return func_name_lut[i].funcName;
        }
    }
    return "unknown";
}


// #define THRESH 1e-4
// __host__ void check_gpu_answer(bElem (*expected)[STRIDE1][STRIDE0], bElem *dev_solution, const char *error_message) {
//     auto solution = (bElem (*)[STRIDE1][STRIDE1]) malloc(STRIDE2 * STRIDE1 * STRIDE0 * sizeof(bElem));
//     gpuMemcpy((bElem *) solution, dev_solution, STRIDE2 * STRIDE1 * STRIDE0 * sizeof(bElem), gpuMemcpyDeviceToHost);

//     for (int i = OFF2; i < N2 + OFF2; i++) {
//         for (int j = OFF1; j < N1 + OFF1; j++) {
//             for (int k = OFF0; k < N0 + OFF0; k++) {
//                 if (abs(solution[i][j][k] - expected[i][j][k]) > THRESH) {
//                     fprintf(stderr, "Got %f, but expected %f at %d %d %d\n", solution[i][j][k], expected[i][j][k], i, j, k);
//                     fflush(stderr);
//                     throw std::runtime_error(error_message);
//                 }
//             }
//         }
//     }

//     free(solution);
// }

// __host__ void check_device_brick(bElem (*expected)[STRIDE1][STRIDE0], BrickStorage device_bstorage, BrickInfo<3> *binfo, unsigned brick_size, unsigned *bgrid, const char *error_message) {
//     auto brick_storage = movBrickStorage(device_bstorage, gpuMemcpyDeviceToHost);
//     BType bOut(binfo, brick_storage, brick_size);
//     if (!compareBrick<3>({N0, N1, N2}, {PADDING0, PADDING1, PADDING2}, {GZ0, GZ1, GZ2}, (bElem *) expected, bgrid, bOut)) {
//         throw std::runtime_error(error_message);
//     }
// }

int main(int argc, char* argv[]) {
    bool verify = true;
    for (int i = 0; i < argc; i++) {
        verify = verify && (strcmp(argv[i], "--nocheck") != 0);
        if (!verify) {
            printf("Not verifying answers!\n");
        }
    }

    // ---- CREATING BASIC ARRAYS ----
    bElem *arr_a = randomArray({STRIDE2, STRIDE1, STRIDE0});
    bElem *arr_b = zeroArray({STRIDE2, STRIDE1, STRIDE0});
    bElem *dev_a;
    bElem *dev_b;
    {
        unsigned size = STRIDE2 * STRIDE1 * STRIDE0 * sizeof(bElem);
        gpuMalloc(&dev_b, size);
        gpuMalloc(&dev_a, size);
        gpuMemcpy(dev_a, arr_a, size, gpuMemcpyHostToDevice);
    }
    // ---- DONE WITH BASIC ARRAYS ----


    // ---- BRICK SETUP ----
    unsigned *bgrid;
    auto binfo = init_grid<3>(bgrid, {NAIVE_BSTRIDE2, NAIVE_BSTRIDE1, NAIVE_BSTRIDE0});
    unsigned *device_bgrid;
    {
        unsigned grid_size = (NAIVE_BSTRIDE0 * NAIVE_BSTRIDE1 * NAIVE_BSTRIDE2) * sizeof(unsigned);
        gpuMalloc(&device_bgrid, grid_size);
        gpuMemcpy(device_bgrid, bgrid, grid_size, gpuMemcpyHostToDevice);
    }
    
    BrickInfo<3> _binfo = movBrickInfo(binfo, gpuMemcpyHostToDevice);
    BrickInfo<3> *device_binfo;
    {
        unsigned binfo_size = sizeof(BrickInfo<3>);
        gpuMalloc(&device_binfo, binfo_size);
        gpuMemcpy(device_binfo, &_binfo, binfo_size, gpuMemcpyHostToDevice);
    }
    auto brick_size = cal_size<BRICK_SIZE>::value;
    // double number of bricks for a and b
    auto brick_storage = BrickStorage::allocate(binfo.nbricks, brick_size * 2);

    BType bIn(&binfo, brick_storage, 0);
    BType bOut(&binfo, brick_storage, brick_size);
    copyToBrick<3>({N0 + 2 * GZ0, N1 + 2 * GZ1, N2 + 2 * GZ2}, {PADDING0, PADDING1, PADDING2}, {0, 0, 0}, arr_a, bgrid, bIn);

    BrickStorage device_bstorage = movBrickStorage(brick_storage, gpuMemcpyHostToDevice);
    bIn = BType(device_binfo, device_bstorage, 0);
    bOut = BType(device_binfo, device_bstorage, brick_size);
    // ---- DONE WITH BRICK SETUP ----

    // ---- RUN TESTS ----
    gpuDeviceSetCacheConfig(gpuFuncCachePreferL1);

    dim3 blocks(BLOCK0, BLOCK1, BLOCK2); 
    dim3 threads(TILE0, TILE1, TILE2);

    // auto expected = (bElem (*)[STRIDE1][STRIDE0]) malloc(STRIDE2 * STRIDE1 * STRIDE0 * sizeof(bElem));
    // {
    //     printf("Generating expected\n");
    //     bElem *dev_gpu_b;
    //     gpuMalloc(&dev_gpu_b, STRIDE2 * STRIDE1 * STRIDE0 * sizeof(bElem));
    //     gpuExecKernel(laplacian_expected_13, 1, 1, (bElem (*)[STRIDE1][STRIDE0]) dev_a, (bElem (*)[STRIDE1][STRIDE0]) dev_gpu_b);
    //     gpuDeviceSynchronize();

    //     gpuMemcpy((bElem *) expected, dev_gpu_b, STRIDE2 * STRIDE1 * STRIDE0 * sizeof(bElem), gpuMemcpyDeviceToHost);
    //     gpuFree(dev_gpu_b);
    // }

    for (int i = 0; i < sfunc_count; i++) {
        setup_funcs[i]();
    }
    for (int i = 0; i < afunc_count; i++) {
        printf("Executing: %s\n", get_atype_kernel_name(array_funcs[i]));
        gpuExecKernel(array_funcs[i], blocks, threads, (bElem (*)[STRIDE1][STRIDE0]) dev_a, (bElem (*)[STRIDE1][STRIDE0]) dev_b);
        gpuSynchronizeAssert();
        // check_gpu_answer(expected, dev_b, "Mismatch!");
        gpuMemcpy(dev_b, arr_b, STRIDE2 * STRIDE1 * STRIDE0 * sizeof(bElem), gpuMemcpyHostToDevice);
    }
    for (int i = 0; i < bfunc_count; i++) {
        printf("Executing: %s\n", get_btype_kernel_name(brick_funcs[i]));
        gpuExecKernel(brick_funcs[i], blocks, VECSIZE, (unsigned (*)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0]) device_bgrid, bIn, bOut);
        gpuSynchronizeAssert();
        // check_device_brick(expected, device_bstorage, &binfo, brick_size, bgrid, "Brick solution mismatch");
        gpuMemcpy(device_bstorage.dat.get(), brick_storage.dat.get(), brick_storage.step * brick_storage.chunks * sizeof(bElem), gpuMemcpyHostToDevice);
    }
    // ---- DONE RUNNING TESTS ----


    // ---- CLEANUP ----
    // free(expected);
    free(arr_a);
    free(arr_b);
    
    free(bgrid);
    free(binfo.adj);
    gpuFree(device_binfo);
    gpuFree(device_bgrid);
    gpuFree(dev_a);
    gpuFree(dev_b);
    return 0;
}