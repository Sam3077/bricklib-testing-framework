#include "hip/hip_runtime.h"

#include <omp.h>
#include "vecscatter.h"
#include "brick.h"

#include "../out/laplacian-stencils.h"

__constant__ bElem dev_coeff[10];

void laplacian_setup() {
    bElem coeff[] = {1.0, 0.9, 0.8, 0.7, 0.6, 0.5, 0.4, 0.3, 0.2, 0.1};
    gpuMemcpyToSymbol(dev_coeff, coeff, sizeof(coeff));
}

__device__ void laplacian_expected_x(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], size_t radius) {
    bElem(*out_sized)[STRIDE1][STRIDE0] = (bElem (*)[STRIDE1][STRIDE0]) out;
    bElem(*in_sized)[STRIDE1][STRIDE0] = (bElem (*)[STRIDE1][STRIDE0]) in;
    for (int k = OFF2; k < N2 + OFF2; k++) {
        for (int j = OFF1; j < N1 + OFF1; j++) {
            for (int i = OFF0; i < N0 + OFF0; i++) {
                out_sized[k][j][i] = dev_coeff[0] * in_sized[k][j][i];
                #pragma unroll
                for (int a = 1; a <= radius; a++) {
                    out_sized[k][j][i] += dev_coeff[a] * (
                        in_sized[k][j][i + a] + in_sized[k][j + a][i] + in_sized[k + a][j][i] +
                        in_sized[k][j][i - a] + in_sized[k][j - a][i] + in_sized[k - a][j][i]
                    );
                }
            }
        }
    }
}

__global__ void laplacian_expected_13(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    return laplacian_expected_x(in, out, 2);
}
__global__ void laplacian_expected_31(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    return laplacian_expected_x(in, out, 5);
}
__global__ void laplacian_expected_49(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    return laplacian_expected_x(in, out, 9);
}

__device__ void naive_xpt_sum(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0], const size_t radius) {
    unsigned i = OFF0 + (blockIdx.x) * TILE0 + threadIdx.x;
    unsigned j = OFF1 + (blockIdx.y) * TILE1 + threadIdx.y;
    unsigned k = OFF2 + (blockIdx.z) * TILE2 + threadIdx.z;

    bElem temp = dev_coeff[0] * in[k][j][i];
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        temp += dev_coeff[a] * (
            in[k][j][i + a] + in[k][j + a][i] + in[k + a][j][i] +
            in[k][j][i - a] + in[k][j - a][i] + in[k - a][j][i]);
    }
    out[k][j][i] = temp;
}

__global__ void laplacian_naive_13(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    return naive_xpt_sum(in, out, 2);
}

__global__ void laplacian_naive_31(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    return naive_xpt_sum(in, out, 5);
}

__global__ void laplacian_naive_49(bElem (*in)[STRIDE1][STRIDE0], bElem (*out)[STRIDE1][STRIDE0]) {
    return naive_xpt_sum(in, out, 8);
}


__device__ void naive_brick_xpt(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType &bIn, BType &bOut, size_t radius) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    unsigned i = threadIdx.x;
    unsigned j = threadIdx.y;
    unsigned k = threadIdx.z;
    bOut[b][k][j][i] = dev_coeff[0] * bIn[b][k][j][i];
    
    #pragma unroll
    for (int a = 1; a <= radius; a++) {
        bOut[b][k][j][i] += dev_coeff[a] * (
            bIn[b][k][j][i + a] + bIn[b][k][j + a][i] + bIn[b][k + a][j][i] +
            bIn[b][k][j][i - a] + bIn[b][k][j - a][i] + bIn[b][k - a][j][i]
        );
    }
}

__global__ void laplacian_naive_bricks_13(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut) {
    naive_brick_xpt(grid, bIn, bOut, 2);
}

__global__ void laplacian_naive_bricks_31(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut) {
    naive_brick_xpt(grid, bIn, bOut, 5);
}

__global__ void laplacian_naive_bricks_49(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut) {
    naive_brick_xpt(grid, bIn, bOut, 8);
}

__global__ void laplacian_codegen_bricks_13(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("13pt.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}

__global__ void laplacian_codegen_bricks_31(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("31pt.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}

__global__ void laplacian_codegen_bricks_49(unsigned (*grid)[NAIVE_BSTRIDE1][NAIVE_BSTRIDE0], BType bIn, BType bOut) {
    unsigned b = grid[blockIdx.z + GB2][blockIdx.y + GB1][blockIdx.x + GB0];
    brick("49pt.py", VSVEC, (TILE2, TILE1, TILE0), (FOLD), b);
}

#define bIn(a, b, c) arr_in[c][b][a]
#define bOut(a, b, c) arr_out[c][b][a]

__global__ void laplacian_codegen_13(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("13pt.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

__global__ void laplacian_codegen_31(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("31pt.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

__global__ void laplacian_codegen_49(bElem (*arr_in)[STRIDE1][STRIDE0], bElem (*arr_out)[STRIDE1][STRIDE0]) {
    long k = OFF2 + (blockIdx.z * TILE2);
    long j = OFF1 + (blockIdx.y * TILE1);
    long i = OFF0 + (blockIdx.x * VECSIZE);
    tile("49pt.py", VSVEC, (TILE2, TILE1, VECSIZE), ("k", "j", "i"), (1, 1, VECSIZE));
}

#undef bIn
#undef bOut